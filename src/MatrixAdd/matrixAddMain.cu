#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include "KernelWrapper.h"
#include "MatrixAddKernel.h"
#include "RoundRobinScheduler.h"
#include "FCFSScheduler.h"
#include "PriorityScheduler.h"

#define NUM_KERNELS 50

hipDevice_t device;
hipCtx_t context;
size_t totalGlobalMem;

void initCuda()
{
    int deviceCount = 0;
    checkCudaErrors(hipInit(0));
    int major = 0, minor = 0;

    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        fprintf(stderr, "Error: no devices supporting CUDA\n");
        exit(-1);
    }

    // get first CUDA device
    checkCudaErrors(hipDeviceGet(&device, 0));
    char name[100];
    hipDeviceGetName(name, 100, device);
    printf("> Using device 0: %s\n", name);

    // get compute capabilities and the devicename
    checkCudaErrors(hipDeviceComputeCapability(&major, &minor, device));
    printf("> GPU Device has SM %d.%d compute capability\n", major, minor);

    checkCudaErrors(hipDeviceTotalMem(&totalGlobalMem, device));
    printf("  Total amount of global memory:   %llu bytes\n",
           (unsigned long long)totalGlobalMem);
    printf("  64-bit Memory Address:           %s\n",
           (totalGlobalMem > (unsigned long long)4 * 1024 * 1024 * 1024L) ? "YES" : "NO");

    checkCudaErrors(hipCtxCreate(&context, 0, device));
}

void finishCuda()
{
    cuCtxDetach(context);
}

int main(int argc, char **argv)
{
    initCuda();
    srand(0);

    // RoundRobinScheduler scheduler;
    // FCFSScheduler scheduler;
    PriorityScheduler scheduler;

    const std::string moduleFile = "matrixAdd.ptx";
    const std::string kernelName = "matrixAdd";

    hipStream_t streams[NUM_KERNELS];
    MatrixAddKernel matrixAddKernels[NUM_KERNELS];
    kernel_attr_t attrs[NUM_KERNELS];
    std::vector<KernelWrapper> wrappers;
    for (int i = 0; i < NUM_KERNELS; ++i)
    {
        checkCudaErrors(hipStreamCreateWithFlags(&streams[i], hipStreamDefault));
        matrixAddKernels[i].getKernelConfig(attrs[i].gridDimX, attrs[i].gridDimY, attrs[i].gridDimZ,
                                            attrs[i].blockDimX, attrs[i].blockDimY, attrs[i].blockDimZ);

        attrs[i].sGridDimX = attrs[i].gridDimX / 16;
        attrs[i].sGridDimY = attrs[i].gridDimY;
        attrs[i].sGridDimZ = attrs[i].gridDimZ;
        attrs[i].sharedMemBytes = 0;
        attrs[i].stream = streams[i];

        KernelWrapper wrapper(&scheduler, context, moduleFile, kernelName, &attrs[i], &matrixAddKernels[i]);
        wrapper.setNiceValue(i % 2);
        wrappers.emplace_back(wrapper);
    }

    struct timeval t0, t1, dt;
    gettimeofday(&t0, NULL);

    scheduler.run();

    for (int i = 0; i < NUM_KERNELS; ++i)
    {
        wrappers[i].launch();
    }

    for (int i = 0; i < NUM_KERNELS; ++i)
    {
        wrappers[i].finish();
    }

    scheduler.stop();
    scheduler.finish();

    gettimeofday(&t1, NULL);
    timersub(&t1, &t0, &dt);
    printf("[main thread] done in %ld.%06ld\n", dt.tv_sec, dt.tv_usec);

    for (int i = 0; i < NUM_KERNELS; ++i)
    {
        checkCudaErrors(hipStreamDestroy(streams[i]));
    }

    finishCuda();

    return 0;
}
