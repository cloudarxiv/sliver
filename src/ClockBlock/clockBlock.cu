#include "hip/hip_runtime.h"
#include "clockBlock.h"

// This is a kernel that does no real work but runs at least for a specified number of clocks
extern "C" __global__ void clockBlock(int blockOffsetX, int blockOffsetY, int blockOffsetZ, long *d_o, long clock_count)
{
    unsigned int start_clock = (unsigned int)clock();

    long clock_offset = 0;

    while (clock_offset < clock_count)
    {
        unsigned int end_clock = (unsigned int)clock();

        // The code below should work like
        // this (thanks to modular arithmetics):
        //
        // clock_offset = (clock_t) (end_clock > start_clock ?
        //                           end_clock - start_clock :
        //                           end_clock + (0xffffffffu - start_clock));
        //
        // Indeed, let m = 2^32 then
        // end - start = end + m - start (mod m).

        clock_offset = (long)(end_clock - start_clock);
    }

    d_o[0] = clock_offset;
}
