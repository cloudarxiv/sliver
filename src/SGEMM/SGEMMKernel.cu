#include "SGEMMKernel.h"

void SGEMMKernel::memAlloc()
{
    readColMajorMatrixFile(params->inpFiles[1], matArow, matAcol, matA);

    readColMajorMatrixFile(params->inpFiles[2], matBcol, matBrow, matBT);

    A_sz = matArow * matAcol * sizeof(float);
    B_sz = matBrow * matBcol * sizeof(float);
    C_sz = matArow * matBcol * sizeof(float);
    matC = std::vector<float>(C_sz);

    checkCudaErrors(hipMalloc(&dA, A_sz));
    checkCudaErrors(hipMalloc(&dB, B_sz));
    checkCudaErrors(hipMalloc(&dC, C_sz));

    args[3] = &dA;
    args[4] = &matArow;
    args[5] = &dB;
    args[6] = &matBcol;
    args[7] = &dC;
    args[8] = &matAcol;
    args[9] = &alpha;
    args[10] = &beta;
}

void SGEMMKernel::memcpyHtoD(const hipStream_t &stream)
{
    checkCudaErrors(hipMemcpyHtoDAsync(dA, &(matA.front()), A_sz, stream));
    checkCudaErrors(hipMemcpyHtoDAsync(dB, &(matBT.front()), B_sz, stream));
}

void SGEMMKernel::memcpyDtoH(const hipStream_t &stream)
{
    checkCudaErrors(hipMemcpyDtoHAsync(&(matC.front()), dC, C_sz, stream));
}

void SGEMMKernel::memFree()
{
    checkCudaErrors(hipFree(dA));
    checkCudaErrors(hipFree(dB));
    checkCudaErrors(hipFree(dC));
}