#include "ClockBlockKernel.h"

void ClockBlockKernel::memAlloc()
{
    h_a = (long *)malloc(sizeof(long));
    checkCudaErrors(hipMalloc(&d_a, sizeof(long)));

    clock_count = KERNEL_TIME * clockRate;

    args[3] = &d_a;
    args[4] = &clock_count;
}

void ClockBlockKernel::memcpyHtoD(const hipStream_t &stream)
{
}

void ClockBlockKernel::memcpyDtoH(const hipStream_t &stream)
{
    checkCudaErrors(hipMemcpyDtoHAsync(h_a, d_a, sizeof(long), stream));
}

void ClockBlockKernel::memFree()
{
    checkCudaErrors(hipFree(d_a));

    free(h_a);
}