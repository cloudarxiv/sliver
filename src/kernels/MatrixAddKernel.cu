#include "MatrixAddKernel.h"

double drand(const double lo = 0.0, const double hi = 1.0) 
{
    return lo + (hi - lo) / RAND_MAX * rand();
}

void MatrixAddKernel::memAlloc()
{
    h_a = (double *)malloc(N * sizeof(double));
    h_b = (double *)malloc(N * sizeof(double));
    h_c = (double *)malloc(N * sizeof(double));

    for (int i = 0; i < N; ++i)
    {
        h_a[i] = drand();
        h_b[i] = drand();
    }

    checkCudaErrors(hipMalloc(&d_a, N * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_b, N * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_c, N * sizeof(double)));

    // for (int i = 0; i < N; ++i)
    // {
    //     printf("[thread id: %ld] %.3f, %.3f\n", pthread_self(), h_a[i], h_b[i]);
    // }

    args[3] = &d_a;
    args[4] = &d_b;
    args[5] = &d_c;
}

void MatrixAddKernel::memcpyHtoD(const hipStream_t &stream)
{
    checkCudaErrors(hipMemcpyHtoDAsync(d_a, h_a, N * sizeof(double), stream));
    checkCudaErrors(hipMemcpyHtoDAsync(d_b, h_b, N * sizeof(double), stream));
}

void MatrixAddKernel::memcpyDtoH(const hipStream_t &stream)
{
    checkCudaErrors(hipMemcpyDtoHAsync(h_c, d_c, N * sizeof(double), stream));
}

void MatrixAddKernel::memFree()
{
    // for (int i = 0; i < N; ++i)
    // {
    //     printf("[thread id: %ld] %.3f, %.3f, %.3f\n", pthread_self(), h_a[i], h_b[i], h_c[i]);
    // }

    checkCudaErrors(hipFree(d_a));
    checkCudaErrors(hipFree(d_b));
    checkCudaErrors(hipFree(d_c));

    free(h_a);
    free(h_b);
    free(h_c);
}