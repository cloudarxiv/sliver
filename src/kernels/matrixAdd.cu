#include "hip/hip_runtime.h"
#include "matrixAdd.h"

extern "C" __global__ void matrixAdd(int blockOffsetX, int blockOffsetY, int blockOffsetZ, double *a, double *b, double *c)
{
    int tid = (blockIdx.x + blockOffsetX) * blockDim.x + threadIdx.x;
    if (tid < N)
    {
        c[tid] = a[tid] + b[tid];
    }
}